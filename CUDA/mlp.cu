#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

// CUDA based sigmoid function
__device__ double sigmoid(double x) {
    return 1.0 / (1.0 + exp(-x));
}

__device__ double sigmoid_derivative(double x) {
    return x * (1.0 - x);
}

__global__ void forward_layer(double* input, double* weights, double* bias, double* output,
                              int input_size, int hidden_size) {
    // Get the block size for the cuda
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Run on needed threads only
    if (idx < hidden_size) {
        output[idx] = bias[idx];
        for (int j = 0; j < input_size; j++) {
            // Note that weights is a 1-D vector
            output[idx] += input[j] * weights[idx * input_size + j];
        }
        output[idx] = sigmoid(output[idx]);
    }
}

__global__ void backward_layer(double* delta, double* delta_all, double* input, double* weights,
                               double* next_weights, double* bias, int input_size, int output_size,
                               double lr, bool is_output_layer) {
    // Get the block size
    int i = blockIdx.x * blockDim.x * threadIdx.x;
    if (i < input_size) {
        // output_size = 1
        // input_size = 2
        double error = 0.0;
        if (is_output_layer) {
            // Get the error based on the activations
            error = delta - input;
        }
        else {
            // Get the error based on the layers
            for (int j = 0; j < output_size; j++) {
                error += delta_all[j] * next_weights[i * output_size + j];
            }
        }

        // update the delta with the error and get new weights
        delta_all[i] = error * sigmoid_derivative(input[i]);
        bias[i] -= delta_all[i] * lr;

        // Run the update equations
        for (int j = 0; j < output_size; j++) {
            weights[i] -= lr * delta_all[j];
        }
    }
}

// Define the MLP structure
class MLP {
    // All the static shit
    int num_layers;
    std::vector<int> layer_sizes;
    std::vector<std::vector<double>> weights;
    std::vector<std::vector<double>> biases;

    // All the pointers
    double *d_input, *d_output;
    std::vector<double*> d_layer_inputs;
    std::vector<double*> d_layer_output;
    std::vector<double*> d_deltas;
    std::vector<double*> d_weights;
    std::vector<double*> d_biases;
    
    public:
        void initialize_layer(int layer){
            srand(static_cast<unsigned>(time(0)));
            int input_size = this->layer_sizes[layer];
            int output_size = this->layer_sizes[layer + 1];

            // Loop and init each memeber
            for (int i = 0; i < input_size * output_size; i++) {
                this->weights[layer][i] = ((double)rand() / RAND_MAX) - 0.5;
            }
            for (int i = 0; i < output_size; i++) {
                this->biases[layer][i] = ((double)rand() / RAND_MAX) - 0.5;
            }
        }

        MLP(const std::vector<int>& layer_sizes) {
            // Init the instance variable
            this->layer_sizes = layer_sizes;
            this->num_layers = layer_sizes.size();

            // Init the other members
            this->weights.resize(this->num_layers - 1);
            this->biases.resize(this->num_layers - 1);
            this->d_weights.resize(this->num_layers - 1);
            this->d_biases.resize(this->num_layers - 1);

            this->d_layer_inputs.resize(this->num_layers);
            this->d_layer_output.resize(this->num_layers);
            this->d_deltas.resize(this->num_layers);


            // Let's init all the members
            for (int i = 0; i < this->num_layers - 1; i++) {
                int in_layer_size = layer_sizes[i];
                int out_layer_size = layer_sizes[i + 1];
                this->weights[i].resize(in_layer_size * out_layer_size);
                this->biases[i].resize(out_layer_size);

                // Allocate the cuda memeory to all the pointers
                hipMalloc(&d_weights[i], weights[i].size() * sizeof(double));
                hipMalloc(&d_biases[i], biases[i].size() * sizeof(double));

                std::cout << in_layer_size << " | " << out_layer_size << " | "  << weights[i].size() << std::endl;

                hipMalloc(&d_layer_inputs[i], in_layer_size * sizeof(double));
                hipMalloc(&d_layer_output[i], out_layer_size * sizeof(double));
                hipMalloc(&d_deltas[i], out_layer_size * sizeof(double));

                // Init a layer that was just established
                initialize_layer(i);

                // Allocate space
                hipMemcpy(d_weights[i], weights[i].data(), weights[i].size() * sizeof(double), hipMemcpyHostToDevice);
                hipMemcpy(d_biases[i], biases[i].data(), biases[i].size() * sizeof(double), hipMemcpyHostToDevice);
            }

            // Few mem allocation at the last
            int input_size = layer_sizes[0];
            int output_size = layer_sizes[num_layers - 1];
            hipMalloc(&d_input, input_size * sizeof(double));
            hipMalloc(&d_output, output_size * sizeof(double));
            hipMalloc(&d_layer_inputs[num_layers - 1], output_size * sizeof(double));
            hipMalloc(&d_layer_output[num_layers - 1], output_size * sizeof(double));
            hipMalloc(&d_deltas[num_layers - 1], output_size * sizeof(double));
        }

        // Define the forward process
        void forward(const std::vector<double>& input, std::vector<double>& output) { 
            // Allocate the memory
            hipMemcpy(d_input, input.data(), input.size() * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_layer_inputs[0], d_input, input.size() * sizeof(double), hipMemcpyDeviceToDevice);

            hipError_t error = hipGetLastError();
            // Loop and do forward pass
            for (int i = 0; i < num_layers - 1; i++) {
                int input_size = layer_sizes[i];
                int hidden_size = layer_sizes[i + 1];

                forward_layer<<<(hidden_size + 255) / 256, 256>>>(d_layer_inputs[i], d_weights[i], d_biases[i],
                                                                  d_layer_output[i], input_size, hidden_size);

                hipDeviceSynchronize();
                hipMemcpy(d_layer_inputs[i + 1], d_layer_output[i], hidden_size * sizeof(double), hipMemcpyDeviceToDevice);
            }

            // Reize the outputss
            output.resize(layer_sizes[num_layers - 1]);
            error = hipMemcpy(output.data(), d_layer_output[num_layers - 2],
                               layer_sizes[num_layers - 1] * sizeof(double), hipMemcpyDeviceToHost);
        }

        void backward(const std::vector<double>& targets, int lr) {
            // Allocate memory
            hipMemcpy(d_output, targets.data(), targets.size() * sizeof(double), hipMemcpyHostToDevice);

            // Backprop the last layer first to get the output deltas
            backward_layer<<<(layer_sizes[num_layers - 1]) + 255 / 256, 256>>>(d_output, d_deltas[num_layers - 1], d_layer_output[num_layers - 1],
                                                                               d_weights[num_layers - 1], d_weights[num_layers - 2],
                                                                               d_biases[num_layers - 1], layer_sizes[num_layers - 2 ],
                                                                               layer_sizes[num_layers - 1], lr, true);

            // Pass through the rest of the layers
            for (int i = num_layers - 2; i >= 0; i--) {
                backward_layer<<<(layer_sizes[i - 1]) + 255 / 256, 256>>>(d_deltas[i], d_deltas[i - 1], d_output,
                                                                          d_weights[i - 1], d_layer_inputs[i - 1],
                                                                          d_biases[i - 1], layer_sizes[i - 2],
                                                                          layer_sizes[i - 1], lr, false);
            }
        }

        ~MLP() {
            for (int i = 0; i < num_layers - 1; ++i) {
                hipFree(d_weights[i]);
                hipFree(d_biases[i]);
                hipFree(d_layer_inputs[i]);
                hipFree(d_layer_output[i]);
                hipFree(d_deltas[i]);
            }
            hipFree(d_input);
            hipFree(d_output);
        }
};

int main() {
    std::cout << "Simple MLP functions......" << std::endl;
    std::cout << "Loading model" << std::endl;
    std::vector<int> layer_sizes = {7, 2, 2, 1};
    std::cout << "Creating model" << std::endl;
    MLP mlp_mod(layer_sizes);

    // Make some dummy input
    std::vector<std::vector<double>> inputs = {
        {1, 0, 0, 1, 1, 1, 1},
        {0, 0, 0, 1, 1, 0, 1},
        {0, 1, 0, 0, 1, 1, 1},
        {0, 0, 0, 1, 0, 0, 1},
        {0, 0, 1, 0, 1, 1, 1},
        {0, 0, 0, 1, 1, 0, 1},
        {1, 0, 0, 1, 1, 1, 1},
    };

    std::vector<std::vector<double>> targets = {
        {0},
        {1},
        {1},
        {1},
        {1},
        {1},
        {0}
    };

    std::vector<double> output;
    for (const auto& input: inputs) {
        mlp_mod.forward(input, output);
        std::cout << "Input: ";
        for (double i: input) {
            std::cout << input[i] << ", ";
        }
        std::cout << " Output: " << output[0] << std::endl;
    }

    return 0;
}
